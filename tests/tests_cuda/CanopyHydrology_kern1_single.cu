#include "hip/hip_runtime.h"
#include <netcdf.h>
#include <array>
#include <sstream>
#include <iterator>
#include <exception>
#include <string>
#include <stdlib.h>
#include <cstring>
#include <vector>
#include <iostream>
#include <iomanip>
#include <fstream>
#include <time.h>
#include "utils.hh"
#include "readers.hh"

#include "CanopyHydrology_decl.hh"


namespace ELM {
namespace Utils {

static const int n_months = 12;
static const int n_pfts = 17;
using MatrixState = MatrixStatic<n_months, n_pfts>;

static const int n_max_times = 31 * 24 * 2; // max days per month times hours per
                                            // day * half hour timestep
using MatrixForc = MatrixStatic<n_max_times,1>;

} // namespace
} // namespace

int main(int argc, char ** argv)
{
  // dimensions
  const int n_months = 12;
  const int n_pfts = 17;
  const int n_max_times = 31 * 24 * 2; // max days per month times hours per
                                       // day * half hour timestep	

  // fixed magic parameters for now
  const int ctype = 1;
  const int ltype = 1;
  const bool urbpoi = false;
  const bool do_capsnow = false;
  const int frac_veg_nosno = 1;
  const double irrig_rate = 0.;
  int n_irrig_steps_left = 0;

  const double dewmx = 0.1;
  const double dtime = 1800.0;

  // phenology state
  ELM::Utils::MatrixState elai;
  ELM::Utils::MatrixState esai;
  ELM::Utils::read_phenology("../links/surfacedataWBW.nc", n_months, n_pfts, 0, elai, esai);

  // forcing state
  ELM::Utils::MatrixForc forc_rain;
  ELM::Utils::MatrixForc forc_snow;
  ELM::Utils::MatrixForc forc_air_temp;
  const int n_times = ELM::Utils::read_forcing("../links/forcing", n_max_times, 6, 1, forc_rain, forc_snow, forc_air_temp);

  double h2ocan = 0.0;
  double qflx_prec_intr = 0.;
  double qflx_irrig = 0.;
  double qflx_prec_grnd = 0.;
  double qflx_snwcp_liq = 0.;
  double qflx_snwcp_ice = 0.;
  double qflx_snow_grnd_patch = 0.;
  double qflx_rain_grnd = 0.;

  hipEvent_t start, stop;
  float time;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  
  // Wait for GPU to finish before accessing on host
 hipDeviceSynchronize();
  std::ofstream soln_file;
  soln_file.open("test_CanopyHydrology_kern1_single.soln");  
  soln_file << "Timestep, forc_rain, h2ocan, qflx_prec_grnd, qflx_prec_intr" << std::endl;
  hipEventRecord(start, 0);

  for(size_t itime = 0; itime < n_times; itime += 1) {
    // note this call puts all precip as rain for testing
    double total_precip = forc_rain[itime][0] + forc_snow[itime][0];
    ELM::CanopyHydrology_Interception<<<1, 256>>>(dtime, total_precip, 0., irrig_rate,
            ltype, ctype, urbpoi, do_capsnow,
            elai[5][7], esai[5][7], dewmx, frac_veg_nosno,
            h2ocan, n_irrig_steps_left,
            qflx_prec_intr, qflx_irrig, qflx_prec_grnd,
            qflx_snwcp_liq, qflx_snwcp_ice,
            qflx_snow_grnd_patch, qflx_rain_grnd);
		
    soln_file << std::setprecision(16) << itime+1 << "\t" << total_precip << "\t" << h2ocan<< "\t" << qflx_prec_grnd << "\t" << qflx_prec_intr << std::endl;
  }
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    std::cout <<"Time for the kernel: "<< time << std::endl;

  // Free memory
//    hipFree(elai);
//    hipFree(esai);
//    hipFree(forc_rain);
//    hipFree(forc_snow);
//    hipFree(forc_air_temp);
  
  return 0;
}
